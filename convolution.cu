#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define KERNEL_RADIUS 2
#define KERNEL_SIZE (2 * KERNEL_RADIUS + 1)
#define BLOCK_SIZE 16

__constant__ float d_kernel[KERNEL_SIZE * KERNEL_SIZE];

__global__ void convolutionKernel(const uint8_t *inputImage,
                                  uint8_t *outputImage, int width, int height,
                                  int bitCount) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) {
    return;
  }

  float sum[3] = {0.0f, 0.0f, 0.0f};
  for (int ky = -KERNEL_RADIUS; ky <= KERNEL_RADIUS; ky++) {
    for (int kx = -KERNEL_RADIUS; kx <= KERNEL_RADIUS; kx++) {
      int pixelX = min(max(x + kx, 0), width - 1);
      int pixelY = min(max(y + ky, 0), height - 1);
      int pixelIndex = (pixelY * width + pixelX) * 3;

      float pixel[3] = {inputImage[pixelIndex], inputImage[pixelIndex + 1],
                        inputImage[pixelIndex + 2]};

      float kernelValue =
          d_kernel[(ky + KERNEL_RADIUS) * KERNEL_SIZE + (kx + KERNEL_RADIUS)];
      for (int c = 0; c < 3; c++) {
        sum[c] += pixel[c] * kernelValue;
      }
    }
  }

  int outputIndex = (y * width + x) * 3;
  for (int c = 0; c < 3; c++) {
    outputImage[outputIndex + c] = static_cast<uint8_t>(sum[c]);
  }
}
